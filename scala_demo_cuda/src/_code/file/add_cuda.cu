#include "hip/hip_runtime.h"
#include <add_cuda.h>
namespace zjx{
__global__ void addwithcuda(int* x, int* y,int *z){
    *z = *x + *y;
    }
int add_ceshi(int m, int n){
    int *x=0;
    int *y=0;
    int *z=0;
    hipMalloc((void**)&x,sizeof(int));
    hipMalloc((void**)&y,sizeof(int));
    hipMalloc((void**)&z,sizeof(int));
    hipMemcpy(x, &m, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(y, &n, sizeof(int), hipMemcpyHostToDevice);
    addwithcuda<<<1, 1>>>(x,y,z);
    int *result=(int* )malloc(sizeof(int));
    hipMemcpy(result, z, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(x);
    hipFree(y);
    hipFree(z);
    return *result;
    
}
}